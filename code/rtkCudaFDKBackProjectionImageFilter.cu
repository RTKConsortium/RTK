#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
/*****************
*  rtk #includes *
*****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaFDKBackProjectionImageFilter.hcu"

/*****************
*  C   #includes *
*****************/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

/*****************
* CUDA #includes *
*****************/
#include <hip/hip_runtime.h>

// T E X T U R E S ////////////////////////////////////////////////////////
texture<float, hipTextureType2DLayered> tex_proj;
texture<float, 3, hipReadModeElementType> tex_proj_3D;

// Constant memory
__constant__ float c_matrices[SLAB_SIZE * 12]; //Can process stacks of at most SLAB_SIZE projections
__constant__ int3 c_projSize;
__constant__ int3 c_vol_size;

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_( S T A R T )_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

__global__
void kernel_fdk(float *dev_vol_in, float *dev_vol_out, unsigned int Blocks_Y)
{
  // CUDA 2.0 does not allow for a 3D grid, which severely
  // limits the manipulation of large 3D arrays of data.  The
  // following code is a hack to bypass this implementation
  // limitation.
  unsigned int blockIdx_z = blockIdx.y / Blocks_Y;
  unsigned int blockIdx_y = blockIdx.y - __umul24(blockIdx_z, Blocks_Y);
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx_y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx_z, blockDim.z) + threadIdx.z;

  if (i >= c_vol_size.x || j >= c_vol_size.y || k >= c_vol_size.z)
    {
    return;
    }

  // Index row major into the volume
  long int vol_idx = i + (j + k*c_vol_size.y)*(c_vol_size.x);

  float3 ip;
  float  voxel_data = 0;

  for (unsigned int proj = 0; proj<c_projSize.z; proj++)
    {
    // matrix multiply
    ip = matrix_multiply(make_float3(i,j,k), &(c_matrices[12*proj]));

    // Change coordinate systems
    ip.z = 1 / ip.z;
    ip.x = ip.x * ip.z;
    ip.y = ip.y * ip.z;

    // Get texture point, clip left to GPU
    voxel_data += tex3D(tex_proj_3D, ip.x, ip.y, proj + 0.5) *  ip.z * ip.z;
    }

  // Place it into the volume
  dev_vol_out[vol_idx] = dev_vol_in[vol_idx] + voxel_data;
}

__global__
void kernel_fdk_3Dgrid(float *dev_vol_in, float * dev_vol_out)
{
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

  if (i >= c_vol_size.x || j >= c_vol_size.y || k >= c_vol_size.z)
    {
    return;
    }

  // Index row major into the volume
  long int vol_idx = i + (j + k*c_vol_size.y)*(c_vol_size.x);

  float3 ip;
  float  voxel_data = 0;

  for (unsigned int proj = 0; proj<c_projSize.z; proj++)
    {
    // matrix multiply
    ip = matrix_multiply(make_float3(i,j,k), &(c_matrices[12*proj]));

    // Change coordinate systems
    ip.z = 1 / ip.z;
    ip.x = ip.x * ip.z;
    ip.y = ip.y * ip.z;

    // Get texture point, clip left to GPU, and accumulate in voxel_data
    voxel_data += tex2DLayered(tex_proj, ip.x, ip.y, proj) *  ip.z * ip.z;
    }

  // Place it into the volume
  dev_vol_out[vol_idx] = dev_vol_in[vol_idx] + voxel_data;
}

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-( E N D )-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_back_project /////////////////////////////
void
CUDA_reconstruct_conebeam(
  int proj_size[3],
  int vol_size[3],
  float *matrices,
  float *dev_vol_in,
  float *dev_vol_out,
  float *dev_proj)
{
  int device;
  hipGetDevice(&device);

  // Copy the size of inputs into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_projSize), proj_size, sizeof(int3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_vol_size), vol_size, sizeof(int3));

  // Copy the projection matrices into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_matrices), &(matrices[0]), 12 * sizeof(float) * proj_size[2]);

  // set texture parameters
  tex_proj.addressMode[0] = hipAddressModeBorder;
  tex_proj.addressMode[1] = hipAddressModeBorder;
  tex_proj.addressMode[2] = hipAddressModeBorder;
  tex_proj.filterMode = hipFilterModeLinear;
  tex_proj.normalized = false; // don't access with normalized texture coords

  tex_proj_3D.addressMode[0] = hipAddressModeBorder;
  tex_proj_3D.addressMode[1] = hipAddressModeBorder;
  tex_proj_3D.addressMode[2] = hipAddressModeBorder;
  tex_proj_3D.filterMode = hipFilterModeLinear;
  tex_proj_3D.normalized = false; // don't access with normalized texture coords

  // Copy projection data to array, bind the array to the texture
  hipExtent projExtent = make_hipExtent(proj_size[0], proj_size[1], proj_size[2]);
  hipArray *array_proj;
  static hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK_ERROR;

  // Allocate array for input projections, in order to bind them to
  // either a 2D layered texture (requires GetCudaComputeCapability >= 2.0) or
  // a 3D texture
  if(CUDA_VERSION<4000 || GetCudaComputeCapability(device).first<=1)
    hipMalloc3DArray((hipArray**)&array_proj, &channelDesc, projExtent);
  else
    hipMalloc3DArray((hipArray**)&array_proj, &channelDesc, projExtent, hipArrayLayered);
  CUDA_CHECK_ERROR;

  // Copy data to 3D array
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr   = make_hipPitchedPtr(dev_proj, proj_size[0]*sizeof(float), proj_size[0], proj_size[1]);
  copyParams.dstArray = (hipArray*)array_proj;
  copyParams.extent   = projExtent;
  copyParams.kind     = hipMemcpyDeviceToDevice;
  hipMemcpy3D(&copyParams);
  CUDA_CHECK_ERROR;

  // Thread Block Dimensions
  const int tBlock_x = 16;
  const int tBlock_y = 4;
  const int tBlock_z = 4;

  // Each element in the volume (each voxel) gets 1 thread
  unsigned int  blocksInX = (vol_size[0]-1)/tBlock_x + 1;
  unsigned int  blocksInY = (vol_size[1]-1)/tBlock_y + 1;
  unsigned int  blocksInZ = (vol_size[2]-1)/tBlock_z + 1;

  // Run kernels. Note: Projection data is passed via texture memory,
  // transform matrix is passed via constant memory
  if(CUDA_VERSION<4000 || GetCudaComputeCapability(device).first<=1)
    {
    // Compute block and grid sizes
    dim3 dimGrid  = dim3(blocksInX, blocksInY*blocksInZ);
    dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

    // Bind the array of projections to a 3D texture
    hipBindTextureToArray(tex_proj_3D, (hipArray*)array_proj, channelDesc);
    CUDA_CHECK_ERROR;

    kernel_fdk <<< dimGrid, dimBlock >>> ( dev_vol_in,
                                           dev_vol_out,
                                           blocksInY );

    // Unbind the image and projection matrix textures
    hipUnbindTexture (tex_proj_3D);
    CUDA_CHECK_ERROR;
    }
  else
    {
    // Compute block and grid sizes
    dim3 dimGrid  = dim3(blocksInX, blocksInY, blocksInZ);
    dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);
    CUDA_CHECK_ERROR;

    // Bind the array of projections to a 2D layered texture
    hipBindTextureToArray(tex_proj, (hipArray*)array_proj, channelDesc);
    CUDA_CHECK_ERROR;

    kernel_fdk_3Dgrid <<< dimGrid, dimBlock >>> ( dev_vol_in,
                                                  dev_vol_out);

    // Unbind the image and projection matrix textures
    hipUnbindTexture (tex_proj);
    CUDA_CHECK_ERROR;
    }

  // Cleanup
  hipFreeArray ((hipArray*)array_proj);
  CUDA_CHECK_ERROR;
}

