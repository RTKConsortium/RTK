#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
/*****************
*  rtk #includes *
*****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaBackProjectionImageFilter.hcu"

/*****************
*  C   #includes *
*****************/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

/*****************
* CUDA #includes *
*****************/
#include <hip/hip_runtime.h>

// T E X T U R E S ////////////////////////////////////////////////////////
texture<float, hipTextureType2DLayered> tex_proj;
texture<float, 3, hipReadModeElementType> tex_proj_3D;

// Constant memory
__constant__ float c_matrices[SLAB_SIZE * 12]; //Can process stacks of at most SLAB_SIZE projections
__constant__ int3 c_projSize;
__constant__ int3 c_volSize;

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_( S T A R T )_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

__global__
void kernel(float *dev_vol_in, float *dev_vol_out, unsigned int Blocks_Y)
{
  // CUDA 2.0 does not allow for a 3D grid, which severely
  // limits the manipulation of large 3D arrays of data.  The
  // following code is a hack to bypass this implementation
  // limitation.
  unsigned int blockIdx_z = blockIdx.y / Blocks_Y;
  unsigned int blockIdx_y = blockIdx.y - __umul24(blockIdx_z, Blocks_Y);
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx_y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx_z, blockDim.z) + threadIdx.z;

  if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z)
    {
    return;
    }

  // Index row major into the volume
  long int vol_idx = i + (j + k*c_volSize.y)*(c_volSize.x);

  float3 ip;
  float  voxel_data = 0;

  for (unsigned int proj = 0; proj<c_projSize.z; proj++)
    {
    // matrix multiply
    ip = matrix_multiply(make_float3(i,j,k), &(c_matrices[12*proj]));

    // Change coordinate systems
    ip.z = 1 / ip.z;
    ip.x = ip.x * ip.z;
    ip.y = ip.y * ip.z;

    // Get texture point, clip left to GPU
    voxel_data += tex3D(tex_proj_3D, ip.x, ip.y, proj + 0.5);
    }

  // Place it into the volume
  dev_vol_out[vol_idx] = dev_vol_in[vol_idx] + voxel_data;
}

__global__
void kernel_cylindrical_detector(float *dev_vol_in, float *dev_vol_out, unsigned int Blocks_Y, double radius)
{
  // CUDA 2.0 does not allow for a 3D grid, which severely
  // limits the manipulation of large 3D arrays of data.  The
  // following code is a hack to bypass this implementation
  // limitation.
  unsigned int blockIdx_z = blockIdx.y / Blocks_Y;
  unsigned int blockIdx_y = blockIdx.y - __umul24(blockIdx_z, Blocks_Y);
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx_y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx_z, blockDim.z) + threadIdx.z;

  if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z)
    {
    return;
    }

  // Index row major into the volume
  long int vol_idx = i + (j + k*c_volSize.y)*(c_volSize.x);

  float3 ip;
  float  voxel_data = 0;

  for (unsigned int proj = 0; proj<c_projSize.z; proj++)
    {
    // matrix multiply
    ip = matrix_multiply(make_float3(i,j,k), &(c_matrices[12*proj]));

    // Change coordinate systems
    ip.z = 1 / ip.z;
    ip.x = ip.x * ip.z;
    ip.y = ip.y * ip.z;

    // Apply correction for cylindrical detector
    double u = ip.y;
    ip.y = radius * atan(u / radius);
    ip.x = ip.x * radius / sqrt(radius * radius + u * u);

    // Get texture point, clip left to GPU
    voxel_data += tex3D(tex_proj_3D, ip.x, ip.y, proj + 0.5);
    }

  // Place it into the volume
  dev_vol_out[vol_idx] = dev_vol_in[vol_idx] + voxel_data;
}

__global__
void kernel_3Dgrid(float *dev_vol_in, float * dev_vol_out)
{
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

  if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z)
    {
    return;
    }

  // Index row major into the volume
  long int vol_idx = i + (j + k*c_volSize.y)*(c_volSize.x);

  float3 ip;
  float  voxel_data = 0;

  for (unsigned int proj = 0; proj<c_projSize.z; proj++)
    {
    // matrix multiply
    ip = matrix_multiply(make_float3(i,j,k), &(c_matrices[12*proj]));

    // Change coordinate systems
    ip.z = 1 / ip.z;
    ip.x = ip.x * ip.z;
    ip.y = ip.y * ip.z;

    // Get texture point, clip left to GPU, and accumulate in voxel_data
    voxel_data += tex2DLayered(tex_proj, ip.x, ip.y, proj);
    }

  // Place it into the volume
  dev_vol_out[vol_idx] = dev_vol_in[vol_idx] + voxel_data;
}

__global__
void kernel_3Dgrid_cylindrical_detector(float *dev_vol_in, float * dev_vol_out, double radius)
{
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

  if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z)
    {
    return;
    }

  // Index row major into the volume
  long int vol_idx = i + (j + k*c_volSize.y)*(c_volSize.x);

  float3 ip;
  float  voxel_data = 0;

  for (unsigned int proj = 0; proj<c_projSize.z; proj++)
    {
    // matrix multiply
    ip = matrix_multiply(make_float3(i,j,k), &(c_matrices[12*proj]));

    // Change coordinate systems
    ip.z = 1 / ip.z;
    ip.x = ip.x * ip.z;
    ip.y = ip.y * ip.z;

    // Apply correction for cylindrical detector
    double u = ip.y;
    ip.y = radius * atan(u / radius);
    ip.x = ip.x * radius / sqrt(radius * radius + u * u);

    // Get texture point, clip left to GPU, and accumulate in voxel_data
    voxel_data += tex2DLayered(tex_proj, ip.x, ip.y, proj);
    }

  // Place it into the volume
  dev_vol_out[vol_idx] = dev_vol_in[vol_idx] + voxel_data;
}

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-( E N D )-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_back_project /////////////////////////////
void
CUDA_back_project(int projSize[3],
  int volSize[3],
  float *matrices,
  float *dev_vol_in,
  float *dev_vol_out,
  float *dev_proj,
  double radiusCylindricalDetector)
{
  int device;
  hipGetDevice(&device);

  // Copy the size of inputs into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_projSize), projSize, sizeof(int3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_volSize), volSize, sizeof(int3));

  // Copy the projection matrices into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_matrices), &(matrices[0]), 12 * sizeof(float) * projSize[2]);

  // set texture parameters
  tex_proj.addressMode[0] = hipAddressModeBorder;
  tex_proj.addressMode[1] = hipAddressModeBorder;
  tex_proj.addressMode[2] = hipAddressModeBorder;
  tex_proj.filterMode = hipFilterModeLinear;
  tex_proj.normalized = false; // don't access with normalized texture coords

  tex_proj_3D.addressMode[0] = hipAddressModeBorder;
  tex_proj_3D.addressMode[1] = hipAddressModeBorder;
  tex_proj_3D.addressMode[2] = hipAddressModeBorder;
  tex_proj_3D.filterMode = hipFilterModeLinear;
  tex_proj_3D.normalized = false; // don't access with normalized texture coords

  // Copy projection data to array, bind the array to the texture
  hipExtent projExtent = make_hipExtent(projSize[0], projSize[1], projSize[2]);
  hipArray *array_proj;
  static hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK_ERROR;

  // Allocate array for input projections, in order to bind them to
  // either a 2D layered texture (requires GetCudaComputeCapability >= 2.0) or
  // a 3D texture
  if(CUDA_VERSION<4000 || GetCudaComputeCapability(device).first<=1)
    hipMalloc3DArray((hipArray**)&array_proj, &channelDesc, projExtent);
  else
    hipMalloc3DArray((hipArray**)&array_proj, &channelDesc, projExtent, hipArrayLayered);
  CUDA_CHECK_ERROR;

  // Copy data to 3D array
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr   = make_hipPitchedPtr(dev_proj, projSize[0]*sizeof(float), projSize[0], projSize[1]);
  copyParams.dstArray = (hipArray*)array_proj;
  copyParams.extent   = projExtent;
  copyParams.kind     = hipMemcpyDeviceToDevice;
  hipMemcpy3D(&copyParams);
  CUDA_CHECK_ERROR;

  // Thread Block Dimensions
  const int tBlock_x = 16;
  const int tBlock_y = 4;
  const int tBlock_z = 4;

  // Each element in the volume (each voxel) gets 1 thread
  unsigned int  blocksInX = (volSize[0]-1)/tBlock_x + 1;
  unsigned int  blocksInY = (volSize[1]-1)/tBlock_y + 1;
  unsigned int  blocksInZ = (volSize[2]-1)/tBlock_z + 1;

  // Run kernels. Note: Projection data is passed via texture memory,
  // transform matrix is passed via constant memory
  if(CUDA_VERSION<4000 || GetCudaComputeCapability(device).first<=1)
    {
    // Compute block and grid sizes
    dim3 dimGrid  = dim3(blocksInX, blocksInY*blocksInZ);
    dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

    // Bind the array of projections to a 3D texture
    hipBindTextureToArray(tex_proj_3D, (hipArray*)array_proj, channelDesc);
    CUDA_CHECK_ERROR;

    if (radiusCylindricalDetector == 0)
      kernel <<< dimGrid, dimBlock >>> ( dev_vol_in,
                                         dev_vol_out,
                                         blocksInY );
    else
      kernel_cylindrical_detector  <<< dimGrid, dimBlock >>> ( dev_vol_in,
                                                               dev_vol_out,
                                                               blocksInY,
                                                               radiusCylindricalDetector);

    // Unbind the image and projection matrix textures
    hipUnbindTexture (tex_proj_3D);
    CUDA_CHECK_ERROR;
    }
  else
    {
    // Compute block and grid sizes
    dim3 dimGrid  = dim3(blocksInX, blocksInY, blocksInZ);
    dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);
    CUDA_CHECK_ERROR;

    // Bind the array of projections to a 2D layered texture
    hipBindTextureToArray(tex_proj, (hipArray*)array_proj, channelDesc);
    CUDA_CHECK_ERROR;

    if (radiusCylindricalDetector == 0)
      kernel_3Dgrid <<< dimGrid, dimBlock >>> ( dev_vol_in, dev_vol_out);
    else
      kernel_3Dgrid_cylindrical_detector <<< dimGrid, dimBlock >>> ( dev_vol_in, dev_vol_out, radiusCylindricalDetector);

    // Unbind the image and projection matrix textures
    hipUnbindTexture (tex_proj);
    CUDA_CHECK_ERROR;
    }

  // Cleanup
  hipFreeArray ((hipArray*)array_proj);
  CUDA_CHECK_ERROR;
}
