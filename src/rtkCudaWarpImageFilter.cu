#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         https://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
/*****************
 *  rtk #includes *
 *****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaWarpImageFilter.hcu"

/*****************
 *  C   #includes *
 *****************/
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>

/*****************
 * CUDA #includes *
 *****************/
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// T E X T U R E S ////////////////////////////////////////////////////////
texture<float, 1, hipReadModeElementType> tex_IndexOutputToPPOutputMatrix;
texture<float, 1, hipReadModeElementType> tex_IndexOutputToIndexDVFMatrix;
texture<float, 1, hipReadModeElementType> tex_PPInputToIndexInputMatrix;

texture<float, 3, hipReadModeElementType> tex_xdvf;
texture<float, 3, hipReadModeElementType> tex_ydvf;
texture<float, 3, hipReadModeElementType> tex_zdvf;
texture<float, 3, hipReadModeElementType> tex_input_vol;
///////////////////////////////////////////////////////////////////////////

// CONSTANTS //////////////////////////////////////////////////////////////
__constant__ float c_IndexOutputToPPOutputMatrix[12];
__constant__ float c_IndexOutputToIndexDVFMatrix[12];
__constant__ float c_PPInputToIndexInputMatrix[12];
////////////////////////////////////////////////////////////////////////////

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_( S T A R T )_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

__global__ void
kernel_3Dgrid(float * dev_vol_out, int3 vol_dim)
{
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

  if (i >= vol_dim.x || j >= vol_dim.y || k >= vol_dim.z)
  {
    return;
  }

  // Index row major into the volume
  long int vol_idx = i + (j + k * vol_dim.y) * (vol_dim.x);

  // Matrix multiply to get the index in the DVF texture of the current point in the output volume
  float3 IndexInDVF = matrix_multiply(make_float3(i, j, k), c_IndexOutputToIndexDVFMatrix);

  // Get each component of the displacement vector by
  // interpolation in the dvf
  float3 Displacement;
  Displacement.x = tex3D(tex_xdvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);
  Displacement.y = tex3D(tex_ydvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);
  Displacement.z = tex3D(tex_zdvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);

  // Matrix multiply to get the physical coordinates of the current point in the output volume
  float3 PP = matrix_multiply(make_float3(i, j, k), c_IndexOutputToPPOutputMatrix);

  // Get the index corresponding to the current physical point in output displaced by the displacement vector
  PP += Displacement;

  // Convert it to a continuous index
  float3 IndexInInput = matrix_multiply(PP, c_PPInputToIndexInputMatrix);

  // Interpolate in the input and copy into the output
  dev_vol_out[vol_idx] = tex3D(tex_input_vol, IndexInInput.x + 0.5f, IndexInInput.y + 0.5f, IndexInInput.z + 0.5f);
}

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-( E N D )-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_warp /////////////////////////////
void
CUDA_warp(int     input_vol_dim[3],
          int     input_dvf_dim[3],
          int     output_vol_dim[3],
          float   IndexOutputToPPOutputMatrix[12],
          float   IndexOutputToIndexDVFMatrix[12],
          float   PPInputToIndexInputMatrix[12],
          float * dev_input_vol,
          float * dev_output_vol,
          float * dev_DVF,
          bool    isLinear)
{
  // Prepare channel description for arrays
  static hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

  // Create CUBLAS context
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  ///////////////////////////////////
  // For each component of the dvf, perform a strided copy (pick every third
  // float from dev_input_dvf) into a 3D array, and bind the array to a 3D texture

  // Extent stuff, will be used for each component extraction
  hipExtent dvfExtent = make_hipExtent(input_dvf_dim[0], input_dvf_dim[1], input_dvf_dim[2]);

  // Set texture parameters
  tex_xdvf.addressMode[0] = hipAddressModeBorder;
  tex_xdvf.addressMode[1] = hipAddressModeBorder;
  tex_xdvf.addressMode[2] = hipAddressModeBorder;
  tex_xdvf.filterMode = hipFilterModeLinear;
  tex_xdvf.normalized = false; // don't access with normalized texture coords

  tex_ydvf.addressMode[0] = hipAddressModeBorder;
  tex_ydvf.addressMode[1] = hipAddressModeBorder;
  tex_ydvf.addressMode[2] = hipAddressModeBorder;
  tex_ydvf.filterMode = hipFilterModeLinear;
  tex_ydvf.normalized = false;

  tex_zdvf.addressMode[0] = hipAddressModeBorder;
  tex_zdvf.addressMode[1] = hipAddressModeBorder;
  tex_zdvf.addressMode[2] = hipAddressModeBorder;
  tex_zdvf.filterMode = hipFilterModeLinear;
  tex_zdvf.normalized = false;

  // Allocate an intermediate memory space to extract x, y and z components of the DVF
  float * DVFcomponent;
  int     numel = input_dvf_dim[0] * input_dvf_dim[1] * input_dvf_dim[2];
  hipMalloc(&DVFcomponent, numel * sizeof(float));
  float one = 1.0;

  // Allocate the arrays used for textures
  hipArray ** DVFcomponentArrays = new hipArray *[3];
  CUDA_CHECK_ERROR;

  // Copy image data to arrays. The tricky part is the make_hipPitchedPtr.
  // The best way to understand it is to read
  // https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
  for (unsigned int component = 0; component < 3; component++)
  {
    // Reset the intermediate memory
    hipMemset((void *)DVFcomponent, 0, numel * sizeof(float));

    // Fill it with the current component
    float * pComponent = dev_DVF + component;
    hipblasSaxpy(handle, numel, &one, pComponent, 3, DVFcomponent, 1);

    // Allocate the hipArray and fill it with the current DVFcomponent
    hipMalloc3DArray((hipArray **)&DVFcomponentArrays[component], &channelDesc, dvfExtent);
    hipMemcpy3DParms CopyParams = hipMemcpy3DParms();
    CopyParams.srcPtr =
      make_hipPitchedPtr(DVFcomponent, input_dvf_dim[0] * sizeof(float), input_dvf_dim[0], input_dvf_dim[1]);
    CopyParams.dstArray = (hipArray *)DVFcomponentArrays[component];
    CopyParams.extent = dvfExtent;
    CopyParams.kind = hipMemcpyDeviceToDevice;
    hipMemcpy3D(&CopyParams);
    CUDA_CHECK_ERROR;
  }

  // Intermediate memory is no longer needed
  hipFree(DVFcomponent);

  // Bind 3D arrays to 3D textures
  hipBindTextureToArray(tex_xdvf, (hipArray *)DVFcomponentArrays[0], channelDesc);
  hipBindTextureToArray(tex_ydvf, (hipArray *)DVFcomponentArrays[1], channelDesc);
  hipBindTextureToArray(tex_zdvf, (hipArray *)DVFcomponentArrays[2], channelDesc);
  CUDA_CHECK_ERROR;

  ///////////////////////////////////
  // Do the same for the input volume

  // Extent stuff
  hipExtent volExtent = make_hipExtent(input_vol_dim[0], input_vol_dim[1], input_vol_dim[2]);

  // Set texture parameters
  tex_input_vol.addressMode[0] = hipAddressModeBorder;
  tex_input_vol.addressMode[1] = hipAddressModeBorder;
  tex_input_vol.addressMode[2] = hipAddressModeBorder;
  tex_input_vol.normalized = false; // don't access with normalized texture coords
  if (isLinear)
    tex_input_vol.filterMode = hipFilterModeLinear;
  else
    tex_input_vol.filterMode = hipFilterModePoint;

  // Allocate the array
  hipArray * array_input_vol;
  hipMalloc3DArray((hipArray **)&array_input_vol, &channelDesc, volExtent);
  CUDA_CHECK_ERROR;

  // Copy image data to array
  hipMemcpy3DParms inputCopyParams = hipMemcpy3DParms();
  inputCopyParams.srcPtr =
    make_hipPitchedPtr(dev_input_vol, input_vol_dim[0] * sizeof(float), input_vol_dim[0], input_vol_dim[1]);
  inputCopyParams.dstArray = (hipArray *)array_input_vol;
  inputCopyParams.extent = volExtent;
  inputCopyParams.kind = hipMemcpyDeviceToDevice;
  hipMemcpy3D(&inputCopyParams);
  CUDA_CHECK_ERROR;

  // Bind 3D arrays to 3D textures
  hipBindTextureToArray(tex_input_vol, (hipArray *)array_input_vol, channelDesc);
  CUDA_CHECK_ERROR;

  // Copy matrices into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(
    c_IndexOutputToPPOutputMatrix), IndexOutputToPPOutputMatrix, 12 * sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(
    c_IndexOutputToIndexDVFMatrix), IndexOutputToIndexDVFMatrix, 12 * sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(
    c_PPInputToIndexInputMatrix), PPInputToIndexInputMatrix, 12 * sizeof(float), 0, hipMemcpyHostToDevice);

  //////////////////////////////////////
  /// Run

  int device;
  hipGetDevice(&device);

  // Thread Block Dimensions
  constexpr int tBlock_x = 16;
  constexpr int tBlock_y = 4;
  constexpr int tBlock_z = 4;

  // Each element in the volume (each voxel) gets 1 thread
  unsigned int blocksInX = (output_vol_dim[0] - 1) / tBlock_x + 1;
  unsigned int blocksInY = (output_vol_dim[1] - 1) / tBlock_y + 1;
  unsigned int blocksInZ = (output_vol_dim[2] - 1) / tBlock_z + 1;

  dim3 dimGrid = dim3(blocksInX, blocksInY, blocksInZ);
  dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

  // Note: the DVF and input image are passed via texture memory
  //-------------------------------------
  kernel_3Dgrid<<<dimGrid, dimBlock>>>(dev_output_vol,
                                       make_int3(output_vol_dim[0], output_vol_dim[1], output_vol_dim[2]));

  CUDA_CHECK_ERROR;

  // Unbind the image and projection matrix textures
  hipUnbindTexture(tex_xdvf);
  hipUnbindTexture(tex_ydvf);
  hipUnbindTexture(tex_zdvf);
  hipUnbindTexture(tex_input_vol);
  CUDA_CHECK_ERROR;

  // Cleanup
  CUDA_CHECK_ERROR;
  hipFreeArray((hipArray *)DVFcomponentArrays[0]);
  hipFreeArray((hipArray *)DVFcomponentArrays[1]);
  hipFreeArray((hipArray *)DVFcomponentArrays[2]);
  hipFreeArray((hipArray *)array_input_vol);
  delete[] DVFcomponentArrays;
  CUDA_CHECK_ERROR;

  // Destroy CUBLAS context
  hipblasDestroy(handle);
}
