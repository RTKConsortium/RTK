/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         https://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

#include "rtkCudaUtilities.hcu"

std::vector<int>
GetListOfCudaDevices()
{
  std::vector<int>      deviceList;
  int                   deviceCount;
  struct hipDeviceProp_t properties;
  if (hipGetDeviceCount(&deviceCount) == hipSuccess)
  {
    for (int device = 0; device < deviceCount; ++device)
    {
      hipGetDeviceProperties(&properties, device);
      if (properties.major != 9999) /* 9999 means emulation only */
        deviceList.push_back(device);
    }
  }
  if (deviceList.size() < 1)
    itkGenericExceptionMacro(<< "No CUDA device available");

  return deviceList;
}

std::pair<int, int>
GetCudaComputeCapability(int device)
{
  struct hipDeviceProp_t properties;
  if (hipGetDeviceProperties(&properties, device) != hipSuccess)
    itkGenericExceptionMacro(<< "Invalid CUDA device");
  return std::make_pair(properties.major, properties.minor);
}

size_t
GetFreeGPUGlobalMemory(int device)
{
  // The return result of cuda utility methods are stored in a hipError_t
  hipError_t result;

  // create cuda context
  hipCtx_t cudaContext;
  result = hipCtxCreate(&cudaContext, hipDeviceScheduleAuto, device);
  if (result != hipSuccess)
  {
    itkGenericExceptionMacro(<< "Could not create context on this CUDA device");
  }

  // get the amount of free memory on the graphics card
  size_t free;
  size_t total;
  result = hipMemGetInfo(&free, &total);
  if (result != hipSuccess)
  {
    itkGenericExceptionMacro(<< "Could not obtain information on free memory on this CUDA device");
  }

  hipCtxDestroy(cudaContext);

  return free;
}

__host__ void
prepareVectorTextureObject(int                                size[3],
                           const float *                      dev_ptr,
                           std::vector<hipArray *> &         componentArrays,
                           const unsigned int                 nComponents,
                           std::vector<hipTextureObject_t> & tex,
                           bool                               isProjections)
{
  componentArrays.resize(nComponents);
  tex.resize(nComponents);

  // Create CUBLAS context
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // create texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  for (int component = 0; component < 3; component++)
  {
    if (isProjections)
      texDesc.addressMode[component] = hipAddressModeBorder;
    else
      texDesc.addressMode[component] = hipAddressModeClamp;
  }
  texDesc.filterMode = hipFilterModeLinear;

  static hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipExtent                   volExtent = make_hipExtent(size[0], size[1], size[2]);

  // Allocate an intermediate memory space to extract the components of the input volume
  float * singleComponent;
  int     numel = size[0] * size[1] * size[2];
  hipMalloc(&singleComponent, numel * sizeof(float));
  CUDA_CHECK_ERROR;
  float one = 1.0;

  // Copy image data to arrays. The tricky part is the make_hipPitchedPtr.
  // The best way to understand it is to read
  // https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
  for (unsigned int component = 0; component < nComponents; component++)
  {
    // Reset the intermediate memory
    hipMemset((void *)singleComponent, 0, numel * sizeof(float));

    // Fill it with the current component
    float * pComponent = dev_ptr + component;
    hipblasSaxpy(handle, numel, &one, pComponent, nComponents, singleComponent, 1);

    // Allocate the hipArray. Projections use layered arrays, volumes use default 3D arrays
    if (isProjections)
      hipMalloc3DArray(&componentArrays[component], &channelDesc, volExtent, hipArrayLayered);
    else
      hipMalloc3DArray(&componentArrays[component], &channelDesc, volExtent);
    CUDA_CHECK_ERROR;

    // Fill it with the current singleComponent
    hipMemcpy3DParms CopyParams = hipMemcpy3DParms();
    CopyParams.srcPtr = make_hipPitchedPtr(singleComponent, size[0] * sizeof(float), size[0], size[1]);
    CopyParams.dstArray = componentArrays[component];
    CopyParams.extent = volExtent;
    CopyParams.kind = hipMemcpyDeviceToDevice;
    hipMemcpy3D(&CopyParams);
    CUDA_CHECK_ERROR;

    // Fill in the texture object with all this information
    resDesc.res.array.array = componentArrays[component];
    hipCreateTextureObject(&tex[component], &resDesc, &texDesc, NULL);
    CUDA_CHECK_ERROR;
  }

  // Intermediate memory is no longer needed
  hipFree(singleComponent);

  // Destroy CUBLAS context
  hipblasDestroy(handle);
}
