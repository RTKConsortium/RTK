/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         https://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

#include "rtkCudaUtilities.hcu"
#include <hipblas.h>

std::vector<int>
GetListOfCudaDevices()
{
  std::vector<int>      deviceList;
  int                   deviceCount;
  struct hipDeviceProp_t properties;
  if (hipGetDeviceCount(&deviceCount) == hipSuccess)
  {
    for (int device = 0; device < deviceCount; ++device)
    {
      hipGetDeviceProperties(&properties, device);
      if (properties.major != 9999) /* 9999 means emulation only */
        deviceList.push_back(device);
    }
  }
  if (deviceList.size() < 1)
    itkGenericExceptionMacro(<< "No CUDA device available");

  return deviceList;
}

std::pair<int, int>
GetCudaComputeCapability(int device)
{
  struct hipDeviceProp_t properties;
  if (hipGetDeviceProperties(&properties, device) != hipSuccess)
    itkGenericExceptionMacro(<< "Invalid CUDA device");
  return std::make_pair(properties.major, properties.minor);
}

size_t
GetFreeGPUGlobalMemory(int device)
{
  // The return result of cuda utility methods are stored in a hipError_t
  hipError_t result;

  // create cuda context
  hipCtx_t cudaContext;
  result = hipCtxCreate(&cudaContext, hipDeviceScheduleAuto, device);
  if (result != hipSuccess)
  {
    itkGenericExceptionMacro(<< "Could not create context on this CUDA device");
  }

  // get the amount of free memory on the graphics card
  size_t free;
  size_t total;
  result = hipMemGetInfo(&free, &total);
  if (result != hipSuccess)
  {
    itkGenericExceptionMacro(<< "Could not obtain information on free memory on this CUDA device");
  }

  hipCtxDestroy(cudaContext);

  return free;
}

__host__ void
prepareScalarTextureObject(int                          size[3],
                           float *                      dev_ptr,
                           hipArray *&                 threeDArray,
                           hipTextureObject_t &        tex,
                           const bool                   isProjections,
                           const bool                   isLinear,
                           const hipTextureAddressMode texAddressMode)
{
  // create texture object
  hipResourceDesc resDesc = {};
  resDesc.resType = hipResourceTypeArray;

  hipTextureDesc texDesc = {};
  texDesc.readMode = hipReadModeElementType;

  for (int component = 0; component < 3; component++)
    texDesc.addressMode[component] = texAddressMode;
  if (isLinear)
    texDesc.filterMode = hipFilterModeLinear;
  else
    texDesc.filterMode = hipFilterModePoint;

  static hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipExtent                   volExtent = make_hipExtent(size[0], size[1], size[2]);

  // Allocate an intermediate memory space to extract the components of the input volume
  float * singleComponent;
  int     numel = size[0] * size[1] * size[2];
  hipMalloc(&singleComponent, numel * sizeof(float));
  CUDA_CHECK_ERROR;

  // Copy image data to arrays. The tricky part is the make_hipPitchedPtr.
  // The best way to understand it is to read
  // https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api

  // Allocate the hipArray. Projections use layered arrays, volumes use default 3D arrays
  if (isProjections)
    hipMalloc3DArray(&threeDArray, &channelDesc, volExtent, hipArrayLayered);
  else
    hipMalloc3DArray(&threeDArray, &channelDesc, volExtent);
  CUDA_CHECK_ERROR;

  // Fill it with the current singleComponent
  hipMemcpy3DParms CopyParams = {};
  CopyParams.srcPtr = make_hipPitchedPtr(dev_ptr, size[0] * sizeof(float), size[0], size[1]);
  CUDA_CHECK_ERROR;
  CopyParams.dstArray = threeDArray;
  CopyParams.extent = volExtent;
  CopyParams.kind = hipMemcpyDeviceToDevice;
  hipMemcpy3D(&CopyParams);
  CUDA_CHECK_ERROR;

  // Fill in the texture object with all this information
  resDesc.res.array.array = threeDArray;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
  CUDA_CHECK_ERROR;
}

__host__ void
prepareVectorTextureObject(int                                size[3],
                           const float *                      dev_ptr,
                           std::vector<hipArray *> &         componentArrays,
                           const unsigned int                 nComponents,
                           std::vector<hipTextureObject_t> & tex,
                           const bool                         isProjections,
                           const hipTextureAddressMode       texAddressMode)
{
  componentArrays.resize(nComponents);
  tex.resize(nComponents);

  // Create CUBLAS context
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // create texture object
  hipResourceDesc resDesc = {};
  resDesc.resType = hipResourceTypeArray;

  hipTextureDesc texDesc = {};
  texDesc.readMode = hipReadModeElementType;
  for (int component = 0; component < 3; component++)
    texDesc.addressMode[component] = texAddressMode;
  texDesc.filterMode = hipFilterModeLinear;

  static hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipExtent                   volExtent = make_hipExtent(size[0], size[1], size[2]);

  // Allocate an intermediate memory space to extract the components of the input volume
  float * singleComponent;
  int     numel = size[0] * size[1] * size[2];
  hipMalloc(&singleComponent, numel * sizeof(float));
  CUDA_CHECK_ERROR;
  float one = 1.0;

  // Copy image data to arrays. The tricky part is the make_hipPitchedPtr.
  // The best way to understand it is to read
  // https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
  for (unsigned int component = 0; component < nComponents; component++)
  {
    // Reset the intermediate memory
    hipMemset((void *)singleComponent, 0, numel * sizeof(float));

    // Fill it with the current component
    const float * pComponent = dev_ptr + component;
    hipblasSaxpy(handle, numel, &one, pComponent, nComponents, singleComponent, 1);

    // Allocate the hipArray. Projections use layered arrays, volumes use default 3D arrays
    if (isProjections)
      hipMalloc3DArray(&componentArrays[component], &channelDesc, volExtent, hipArrayLayered);
    else
      hipMalloc3DArray(&componentArrays[component], &channelDesc, volExtent);
    CUDA_CHECK_ERROR;

    // Fill it with the current singleComponent
    hipMemcpy3DParms CopyParams = hipMemcpy3DParms();
    CopyParams.srcPtr = make_hipPitchedPtr(singleComponent, size[0] * sizeof(float), size[0], size[1]);
    CUDA_CHECK_ERROR;
    CopyParams.dstArray = componentArrays[component];
    CopyParams.extent = volExtent;
    CopyParams.kind = hipMemcpyDeviceToDevice;
    hipMemcpy3D(&CopyParams);
    CUDA_CHECK_ERROR;

    // Fill in the texture object with all this information
    resDesc.res.array.array = componentArrays[component];
    hipCreateTextureObject(&tex[component], &resDesc, &texDesc, NULL);
    CUDA_CHECK_ERROR;
  }

  // Intermediate memory is no longer needed
  hipFree(singleComponent);

  // Destroy CUBLAS context
  hipblasDestroy(handle);
}

__host__ void
prepareGeometryTextureObject(int                   length,
                             const float *         geometry,
                             float *&              dev_geom,
                             hipTextureObject_t & tex_geom,
                             const unsigned int    nParam)
{
  // copy geometry matrix to device, bind the matrix to the texture
  hipMalloc((void **)&dev_geom, length * nParam * sizeof(float));
  CUDA_CHECK_ERROR;
  hipMemcpy(dev_geom, geometry, length * nParam * sizeof(float), hipMemcpyHostToDevice);
  CUDA_CHECK_ERROR;

  // create texture object
  hipResourceDesc resDesc = {};
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = dev_geom;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = 32; // bits per channel
  resDesc.res.linear.sizeInBytes = length * nParam * sizeof(float);

  hipTextureDesc texDesc = {};
  texDesc.readMode = hipReadModeElementType;

  hipCreateTextureObject(&tex_geom, &resDesc, &texDesc, NULL);
  CUDA_CHECK_ERROR;
}
