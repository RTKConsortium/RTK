#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         https://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

#include "rtkCudaLagCorrectionImageFilter.hcu"
#include "rtkCudaUtilities.hcu"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

__constant__ float cst_coef[9];

__global__ void
kernel_lag_correction(int3             proj_idx_in,
                      int3             proj_size_in,
                      int2             proj_size_in_buf,
                      int3             proj_idx_out,
                      int3             proj_size_out,
                      int2             proj_size_out_buf,
                      unsigned short * dev_proj_in,
                      unsigned short * dev_proj_out,
                      float *          state)
{
  constexpr int modelOrder = 4;

  // compute thread index
  int3 tIdx;
  tIdx.x = blockIdx.x * blockDim.x + threadIdx.x;
  tIdx.y = blockIdx.y * blockDim.y + threadIdx.y;
  tIdx.z = blockIdx.z * blockDim.z + threadIdx.z;
  long int tIdx_comp = tIdx.x + tIdx.y * proj_size_out.x + tIdx.z * proj_size_out_buf.x * proj_size_out_buf.y;

  // check if outside of projection grid
  if (tIdx.x >= proj_size_out.x || tIdx.y >= proj_size_out.y || tIdx.z >= proj_size_out.z)
    return;

  // compute projection index from thread index
  int3 pIdx = make_int3(tIdx.x + proj_idx_out.x, tIdx.y + proj_idx_out.y, tIdx.z + proj_idx_out.z);
  // combined proj. index -> use thread index in z because accessing memory only with this index
  long int pIdx_comp = (pIdx.x - proj_idx_in.x) + (pIdx.y - proj_idx_in.y) * proj_size_in_buf.x +
                       (pIdx.z - proj_idx_in.z) * proj_size_in_buf.x * proj_size_in_buf.y;

  long int sIdx_comp = tIdx.x + tIdx.y * proj_size_out.x;
  unsigned idx_s = sIdx_comp * modelOrder;

  float yk = static_cast<float>(dev_proj_in[pIdx_comp]);
  float xk = yk;

  float Sa[modelOrder];
  for (unsigned int n = 0; n < modelOrder; n++)
  {
    // Compute the update of internal state for nth exponential
    float expmA_n = cst_coef[4 + n];
    Sa[n] = expmA_n * state[idx_s + n];

    // Update x[k] by removing contribution of the nth exponential
    float B_n = cst_coef[n];
    xk -= B_n * Sa[n];
  }

  // Apply normalization factor
  xk = xk / cst_coef[8];

  // Update internal state Snk
  for (unsigned int n = 0; n < modelOrder; n++)
  {
    state[idx_s + n] = xk + Sa[n];
  }

  // Avoid negative values
  xk = (xk < 0.0f) ? 0.f : xk;

  dev_proj_out[tIdx_comp] = static_cast<unsigned short>(xk);
}

void
CUDA_lag_correction(int              proj_idx_in[3],      // overlapping input region index
                    int              proj_dim_in[3],      // overlapping input region size
                    int              proj_dim_in_buf[2],  // input size of buffered region
                    int              proj_idx_out[3],     // output region index
                    int              proj_dim_out[3],     // output region size
                    int              proj_dim_out_buf[2], // output size of buffered region
                    unsigned short * dev_proj_in,
                    unsigned short * dev_proj_out,
                    float *          h_state,
                    size_t           state_size,
                    float *          coefficients)
{
  // Thread Block Dimensions
  int tBlock_x = 16;
  int tBlock_y = 16;
  int tBlock_z = 2;

  // Each element in the volume (each voxel) gets 1 thread
  unsigned int blocksInX = (proj_dim_out[0] - 1) / tBlock_x + 1;
  unsigned int blocksInY = (proj_dim_out[1] - 1) / tBlock_y + 1;
  unsigned int blocksInZ = (proj_dim_out[2] - 1) / tBlock_z + 1;

  float * d_state; // device state
  hipMalloc((void **)&d_state, state_size);
  if (hipMemcpy(d_state, h_state, state_size, hipMemcpyHostToDevice) != hipSuccess)
  {
    itkGenericExceptionMacro("Error allocating state");
  }

  if (coefficients[8] <= 0.0)
    coefficients[8] = 1.0;

  hipMemcpyToSymbol(HIP_SYMBOL(cst_coef), coefficients, 9 * sizeof(float));

  dim3 dimGrid = dim3(blocksInX, blocksInY, blocksInZ);
  dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);
  kernel_lag_correction<<<dimGrid, dimBlock>>>(make_int3(proj_idx_in[0], proj_idx_in[1], proj_idx_in[2]),
                                               make_int3(proj_dim_in[0], proj_dim_in[1], proj_dim_in[2]),
                                               make_int2(proj_dim_in_buf[0], proj_dim_in_buf[1]),
                                               make_int3(proj_idx_out[0], proj_idx_out[1], proj_idx_out[2]),
                                               make_int3(proj_dim_out[0], proj_dim_out[1], proj_dim_out[2]),
                                               make_int2(proj_dim_out_buf[0], proj_dim_out_buf[1]),
                                               dev_proj_in,
                                               dev_proj_out,
                                               d_state);

  hipMemcpy(h_state, d_state, state_size, hipMemcpyDeviceToHost);
  hipFree(d_state);

  CUDA_CHECK_ERROR;
}
