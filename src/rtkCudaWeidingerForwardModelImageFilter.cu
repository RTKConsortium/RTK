#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         https://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
/*****************
 *  rtk #includes *
 *****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaWeidingerForwardModelImageFilter.hcu"

/*****************
 *  C   #includes *
 *****************/
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>

/*****************
 * CUDA #includes *
 *****************/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define IDX2D(r, c, cols) ((r) * (cols) + (c))

// CONSTANTS //////////////////////////////////////////////////////////////
__constant__ int3  c_projSize;
__constant__ float c_materialAttenuations[3 * 150];
__constant__ float c_binnedDetectorResponse[5 * 150];
////////////////////////////////////////////////////////////////////////////

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_( S T A R T )_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

template <unsigned int VBins, unsigned int VEnergies, unsigned int VMaterials>
__global__ void
kernel_forward_model(float *      pMatProj,
                     float *      pPhoCount,
                     float *      pSpectrum,
                     float *      pProjOnes,
                     float *      pOut1,
                     float *      pOut2,
                     unsigned int nProjSpectrum,
                     int          nIdxProj)
{
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

  if (i >= c_projSize.x || j >= c_projSize.y || k >= c_projSize.z)
  {
    return;
  }

  // Index row major in the projection
  long int first_proj_idx =
    i + (j + (nIdxProj + k) % nProjSpectrum * c_projSize.y) * c_projSize.x; // To determine the efficient spectrum
  long int proj_idx = i + (j + k * c_projSize.y) * (c_projSize.x);          // For all the rest

  // Compute the efficient spectrum at the current pixel
  float efficientSpectrum[VBins * VEnergies];
  for (unsigned int b = 0; b < VBins; b++)
    for (unsigned int e = 0; e < VEnergies; e++)
      efficientSpectrum[IDX2D(b, e, VEnergies)] =
        pSpectrum[e + VEnergies * first_proj_idx] * c_binnedDetectorResponse[IDX2D(b, e, VEnergies)];

  // Get attenuation factors at each energy from material projections
  float attenuationFactors[VEnergies];
  matrix_matrix_multiply(
    c_materialAttenuations, (float *)&pMatProj[proj_idx * VMaterials], attenuationFactors, VEnergies, 1, VMaterials);

  for (unsigned int e = 0; e < VEnergies; e++)
    attenuationFactors[e] = std::exp(-attenuationFactors[e]);

  // Get the expected photon counts through these attenuations
  float expectedCounts[VBins];
  matrix_matrix_multiply(efficientSpectrum, attenuationFactors, expectedCounts, VBins, 1, VEnergies);

  // Get intermediate variables used in the computation of the first output
  float oneMinusRatios[VBins];
  for (unsigned int b = 0; b < VBins; b++)
    oneMinusRatios[b] = 1 - (pPhoCount[proj_idx * VBins + b] / expectedCounts[b]);

  // Form an intermediate variable used for the gradient of the cost function,
  // (the derivation of the exponential implies that a m_MaterialAttenuations
  // gets out), by equivalent of element-wise product with implicit extension
  float intermForGradient[VEnergies * VMaterials];
  for (unsigned int e = 0; e < VEnergies; e++)
    for (unsigned int m = 0; m < VMaterials; m++)
      intermForGradient[IDX2D(e, m, VMaterials)] =
        c_materialAttenuations[IDX2D(e, m, VMaterials)] * attenuationFactors[e];

  // Multiply by the spectrum
  float interm2ForGradient[VBins * VMaterials];
  matrix_matrix_multiply(efficientSpectrum, intermForGradient, interm2ForGradient, VBins, VMaterials, VEnergies);

  // Take the opposite
  for (unsigned int b = 0; b < VBins; b++)
    for (unsigned int m = 0; m < VMaterials; m++)
      interm2ForGradient[IDX2D(b, m, VMaterials)] *= -1;

  // Compute the product with oneMinusRatios, with implicit extension
  for (unsigned int b = 0; b < VBins; b++)
    for (unsigned int m = 0; m < VMaterials; m++)
      interm2ForGradient[IDX2D(b, m, VMaterials)] *= oneMinusRatios[b];

  // Finally, compute the vector to be written in first output
  // by summing on the bins
  for (unsigned int b = 0; b < VBins; b++)
    for (unsigned int m = 0; m < VMaterials; m++)
      pOut1[proj_idx * VMaterials + m] += interm2ForGradient[IDX2D(b, m, VMaterials)];

  // Now compute output2

  // Form an intermediate variable used for the hessian of the cost function,
  // (the double derivation of the exponential implies that a m_MaterialAttenuations^2
  // gets out), by equivalent of element-wise product with implicit extension
  float intermForHessian[VEnergies * VMaterials * VMaterials];
  for (unsigned int r = 0; r < VEnergies; r++)
    for (unsigned int c = 0; c < VMaterials; c++)
      for (unsigned int c2 = 0; c2 < VMaterials; c2++)
        intermForHessian[(r * VMaterials + c) * VMaterials + c2] = c_materialAttenuations[c + VMaterials * r] *
                                                                   c_materialAttenuations[c2 + VMaterials * r] *
                                                                   attenuationFactors[r];

  // Multiply by the spectrum
  float interm2ForHessian[VBins * VMaterials * VMaterials];
  matrix_matrix_multiply(
    efficientSpectrum, intermForHessian, interm2ForHessian, VBins, VMaterials * VMaterials, VEnergies);

  // Sum on the bins
  for (unsigned int b = 0; b < VBins; b++)
    for (unsigned int c = 0; c < VMaterials * VMaterials; c++)
      pOut2[proj_idx * VMaterials * VMaterials + c] += interm2ForHessian[IDX2D(b, c, VMaterials * VMaterials)];

  // Multiply by the projection of ones
  for (unsigned int c = 0; c < VMaterials * VMaterials; c++)
    pOut2[proj_idx * VMaterials * VMaterials + c] *= pProjOnes[proj_idx];
}

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-( E N D )-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_WeidingerForwardModel /////////////////////////////
void
CUDA_WeidingerForwardModel(int          projectionSize[3],
                           float *      materialAttenuations,
                           float *      binnedDetectorResponse,
                           float *      pMatProj,
                           float *      pPhoCount,
                           float *      pSpectrum,
                           float *      pProjOnes,
                           float *      pOut1,
                           float *      pOut2,
                           unsigned int nBins,
                           unsigned int nEnergies,
                           unsigned int nMaterials,
                           unsigned int nProjSpectrum,
                           int          nIdxProj)
{
  hipMemcpyToSymbol(HIP_SYMBOL(c_projSize), projectionSize, sizeof(int3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_binnedDetectorResponse), &(binnedDetectorResponse[0]), nBins * nEnergies * sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(c_materialAttenuations), &(materialAttenuations[0]), nMaterials * nEnergies * sizeof(float));

  // Set both outputs to zeros
  hipMemset((void *)pOut1, 0, projectionSize[0] * projectionSize[1] * projectionSize[2] * nMaterials * sizeof(float));
  hipMemset((void *)pOut2,
             0,
             projectionSize[0] * projectionSize[1] * projectionSize[2] * nMaterials * nMaterials * sizeof(float));

  dim3 dimBlock = dim3(4, 4, 4);
  dim3 dimGrid = dim3(iDivUp(projectionSize[0], dimBlock.x),
                      iDivUp(projectionSize[1], dimBlock.y),
                      iDivUp(projectionSize[2], dimBlock.z));

  if (nBins == 5 && nEnergies == 150)
  {
    switch (nMaterials)
    {
      case 2:
        kernel_forward_model<5, 150, 2>
          <<<dimGrid, dimBlock>>>(pMatProj, pPhoCount, pSpectrum, pProjOnes, pOut1, pOut2, nProjSpectrum, nIdxProj);
        break;

      case 3:
        kernel_forward_model<5, 150, 3>
          <<<dimGrid, dimBlock>>>(pMatProj, pPhoCount, pSpectrum, pProjOnes, pOut1, pOut2, nProjSpectrum, nIdxProj);
        break;

      default:
      {
        itkGenericExceptionMacro(<< "The CUDA version of WeidingerForwardModel works with hard-coded parameters, "
                                    "currently set to 2 or 3 materials."
                                 << nMaterials << " is not supported.");
      }
    }
    CUDA_CHECK_ERROR;
  }
  else if (nBins == 1 && nEnergies == 79)
  {
    switch (nMaterials)
    {
      case 2:
        kernel_forward_model<1, 79, 2>
          <<<dimGrid, dimBlock>>>(pMatProj, pPhoCount, pSpectrum, pProjOnes, pOut1, pOut2, nProjSpectrum, nIdxProj);
        break;
      case 3:
        kernel_forward_model<1, 79, 3>
          <<<dimGrid, dimBlock>>>(pMatProj, pPhoCount, pSpectrum, pProjOnes, pOut1, pOut2, nProjSpectrum, nIdxProj);
        break;
      default:
      {
        itkGenericExceptionMacro(<< "The CUDA version of WeidingerForwardModel works with hard-coded parameters, "
                                    "currently set to 2 or 3 materials."
                                 << nMaterials << " is not supported.");
      }
    }
    CUDA_CHECK_ERROR;
  }
  else
  {
    itkGenericExceptionMacro(<< "The CUDA version of WeidingerForwardModel works with hard-coded parameters "
                                "(nBins,nEnergies) equal to (5,150),(5,150),(1,79),(1,79).");
  }
  CUDA_CHECK_ERROR;
}
