#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         https://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
/*****************
 *  rtk #includes *
 *****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaFDKBackProjectionImageFilter.hcu"

/*****************
 *  C   #includes *
 *****************/
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>

/*****************
 * CUDA #includes *
 *****************/
#include <hip/hip_runtime.h>

// Constant memory
__constant__ float c_matrices[SLAB_SIZE * 12]; // Can process stacks of at most SLAB_SIZE projections
__constant__ int3 c_projSize;
__constant__ int3 c_vol_size;

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_( S T A R T )_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

__global__ void
kernel_fdk_3Dgrid(float * dev_vol_in, float * dev_vol_out, hipTextureObject_t tex_proj)
{
  itk::SizeValueType i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  itk::SizeValueType j = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  itk::SizeValueType k = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

  if (i >= c_vol_size.x || j >= c_vol_size.y || k >= c_vol_size.z)
  {
    return;
  }

  // Index row major into the volume
  itk::SizeValueType vol_idx = i + (j + k * c_vol_size.y) * (c_vol_size.x);

  float3 ip;
  float  voxel_data = 0;

  for (unsigned int proj = 0; proj < c_projSize.z; proj++)
  {
    // matrix multiply
    ip = matrix_multiply(make_float3(i, j, k), &(c_matrices[12 * proj]));

    // Change coordinate systems
    ip.z = 1 / ip.z;
    ip.x = ip.x * ip.z;
    ip.y = ip.y * ip.z;

    // Get texture point, clip left to GPU, and accumulate in voxel_data
    voxel_data += tex2DLayered<float>(tex_proj, ip.x, ip.y, proj) * ip.z * ip.z;
  }

  // Place it into the volume
  dev_vol_out[vol_idx] = dev_vol_in[vol_idx] + voxel_data;
}

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-( E N D )-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_back_project /////////////////////////////
void
CUDA_reconstruct_conebeam(int     proj_size[3],
                          int     vol_size[3],
                          float * matrices,
                          float * dev_vol_in,
                          float * dev_vol_out,
                          float * dev_proj)
{
  // Copy the size of inputs into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_projSize), proj_size, sizeof(int3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_vol_size), vol_size, sizeof(int3));

  // Copy the projection matrices into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_matrices), &(matrices[0]), 12 * sizeof(float) * proj_size[2]);

  // Thread Block Dimensions
  constexpr int tBlock_x = 16;
  constexpr int tBlock_y = 4;
  constexpr int tBlock_z = 4;

  // Each element in the volume (each voxel) gets 1 thread
  unsigned int blocksInX = (vol_size[0] - 1) / tBlock_x + 1;
  unsigned int blocksInY = (vol_size[1] - 1) / tBlock_y + 1;
  unsigned int blocksInZ = (vol_size[2] - 1) / tBlock_z + 1;

  // Run kernels. Note: Projection data is passed via texture memory,
  // transform matrix is passed via constant memory

  // Compute block and grid sizes
  dim3 dimGrid = dim3(blocksInX, blocksInY, blocksInZ);
  dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

  hipArray *         array_proj;
  hipTextureObject_t tex_proj;
  prepareScalarTextureObject(proj_size, dev_proj, array_proj, tex_proj, true);
  kernel_fdk_3Dgrid<<<dimGrid, dimBlock>>>(dev_vol_in, dev_vol_out, tex_proj);

  // Cleanup
  hipFreeArray((hipArray *)array_proj);
  CUDA_CHECK_ERROR;
  hipDestroyTextureObject(tex_proj);
  CUDA_CHECK_ERROR;
}
