#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         https://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

/*****************
 *  rtk #includes *
 *****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaIntersectBox.hcu"
#include "rtkCudaForwardProjectionImageFilter.hcu"

/*****************
 *  C   #includes *
 *****************/
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>

/*****************
 * CUDA #includes *
 *****************/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CONSTANTS //
__constant__ int3   c_projSize;
__constant__ float3 c_boxMin;
__constant__ float3 c_boxMax;
__constant__ float3 c_spacing;
__constant__ int3   c_volSize;
__constant__ float  c_tStep;
__constant__ float  c_radius;
__constant__ float
  c_translatedProjectionIndexTransformMatrices[SLAB_SIZE * 12]; // Can process stacks of at most SLAB_SIZE projections
__constant__ float
  c_translatedVolumeTransformMatrices[SLAB_SIZE * 12]; // Can process stacks of at most SLAB_SIZE projections
__constant__ float c_sourcePos[SLAB_SIZE * 3];         // Can process stacks of at most SLAB_SIZE projections


//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_( S T A R T )_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

// KERNEL kernel_forwardProject
template <unsigned int VVectorLength>
__global__ void
kernel_forwardProject(float * dev_proj_in, float * dev_proj_out, hipTextureObject_t * dev_tex_vol)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int numThread = j * c_projSize.x + i;

  if (i >= c_projSize.x || j >= c_projSize.y)
    return;

  // Declare variables used in the loop
  Ray    ray;
  float3 pixelPos;
  float  tnear, tfar;

  for (unsigned int proj = 0; proj < c_projSize.z; proj++)
  {
    // Setting ray origin
    ray.o = make_float3(c_sourcePos[3 * proj], c_sourcePos[3 * proj + 1], c_sourcePos[3 * proj + 2]);

    if (c_radius == 0)
    {
      pixelPos = matrix_multiply(make_float3(i, j, 0), &(c_translatedProjectionIndexTransformMatrices[12 * proj]));
    }
    else
    {
      float3 posProj;
      posProj = matrix_multiply(make_float3(i, j, 0), &(c_translatedProjectionIndexTransformMatrices[12 * proj]));
      double a = posProj.x / c_radius;
      posProj.x = sin(a) * c_radius;
      posProj.z += (1. - cos(a)) * c_radius;
      pixelPos = matrix_multiply(posProj, &(c_translatedVolumeTransformMatrices[12 * proj]));
    }

    ray.d = pixelPos - ray.o;
    ray.d = ray.d / sqrtf(dot(ray.d, ray.d));

    int projOffset = numThread + proj * c_projSize.x * c_projSize.y;

    // Detect intersection with box
    if (!intersectBox(ray, &tnear, &tfar, c_boxMin, c_boxMax) || tfar < 0.f)
    {
      for (unsigned int c = 0; c < VVectorLength; c++)
        dev_proj_out[projOffset * VVectorLength + c] = dev_proj_in[projOffset * VVectorLength + c];
    }
    else
    {
      if (tnear < 0.f)
        tnear = 0.f; // clamp to near plane

      // Step length in mm
      float3 dirInMM = c_spacing * ray.d;
      float  vStep = c_tStep / sqrtf(dot(dirInMM, dirInMM));
      float3 step = vStep * ray.d;

      // First position in the box
      float3 pos;
      float  halfVStep = 0.5f * vStep;
      tnear = tnear + halfVStep;
      pos = ray.o + tnear * ray.d;

      float t;
      float sample[VVectorLength];
      float sum[VVectorLength];
      for (unsigned int c = 0; c < VVectorLength; c++)
      {
        sample[c] = 0.0f;
        sum[c] = 0.0f;
      }

      for (t = tnear; t <= tfar; t += vStep)
      {
        // Read from 3D texture from volume(s)
        for (unsigned int c = 0; c < VVectorLength; c++)
          sample[c] = tex3D<float>(dev_tex_vol[c], pos.x, pos.y, pos.z);

        // Accumulate
        for (unsigned int c = 0; c < VVectorLength; c++)
          sum[c] += sample[c];

        // Step forward
        pos += step;
      }

      // Update the output projection pixels
      for (unsigned int c = 0; c < VVectorLength; c++)
        dev_proj_out[projOffset * VVectorLength + c] =
          dev_proj_in[projOffset * VVectorLength + c] + (sum[c] + (tfar - t + halfVStep) / vStep * sample[c]) * c_tStep;
    }
  }
}

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-( E N D )-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_forward_project() //////////////////////////////////
void
CUDA_forward_project(int          projSize[3],
                     int          volSize[3],
                     float *      translatedProjectionIndexTransformMatrices,
                     float *      translatedVolumeTransformMatrices,
                     float *      dev_proj_in,
                     float *      dev_proj_out,
                     float *      dev_vol,
                     float        t_step,
                     float *      source_positions,
                     float        radiusCylindricalDetector,
                     float        box_min[3],
                     float        box_max[3],
                     float        spacing[3],
                     unsigned int vectorLength)
{
  // Constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_projSize), projSize, sizeof(int3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_boxMin), box_min, sizeof(float3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_boxMax), box_max, sizeof(float3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_spacing), spacing, sizeof(float3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_volSize), volSize, sizeof(int3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_tStep), &t_step, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(c_radius), &radiusCylindricalDetector, sizeof(float));

  dim3 dimBlock = dim3(16, 16, 1);
  dim3 dimGrid = dim3(iDivUp(projSize[0], dimBlock.x), iDivUp(projSize[1], dimBlock.x));

  // Copy the source position matrix into a float3 in constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_sourcePos), &(source_positions[0]), 3 * sizeof(float) * projSize[2]);

  // Copy the projection matrices into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_translatedProjectionIndexTransformMatrices),
                     &(translatedProjectionIndexTransformMatrices[0]),
                     12 * sizeof(float) * projSize[2]);
  hipMemcpyToSymbol(HIP_SYMBOL(
    c_translatedVolumeTransformMatrices), &(translatedVolumeTransformMatrices[0]), 12 * sizeof(float) * projSize[2]);

  // Prepare texture objects
  std::vector<hipArray *>         volComponentArrays;
  std::vector<hipTextureObject_t> tex_vol;
  prepareVectorTextureObject(volSize, dev_vol, volComponentArrays, vectorLength, tex_vol, false, hipAddressModeClamp);

  // Copy them to a device pointer, since it will have to be de-referenced in the kernels
  hipTextureObject_t * dev_tex_vol;
  hipMalloc(&dev_tex_vol, vectorLength * sizeof(hipTextureObject_t));
  hipMemcpy(dev_tex_vol, tex_vol.data(), vectorLength * sizeof(hipTextureObject_t), hipMemcpyHostToDevice);

  // Run the kernel. Since "vectorLength" is passed as a function argument, not as a template argument,
  // the compiler can't assume it's constant, and a dirty trick has to be used.
  // I did not manage to make CUDA_forward_project templated over vectorLength,
  // which would be the best solution
  switch (vectorLength)
  {
    case 1:
      kernel_forwardProject<1><<<dimGrid, dimBlock>>>(dev_proj_in, dev_proj_out, dev_tex_vol);
      break;

    case 2:
      kernel_forwardProject<2><<<dimGrid, dimBlock>>>(dev_proj_in, dev_proj_out, dev_tex_vol);
      break;

    case 3:
      kernel_forwardProject<3><<<dimGrid, dimBlock>>>(dev_proj_in, dev_proj_out, dev_tex_vol);
      break;

    default:
    {
      itkGenericExceptionMacro("Vector length " << vectorLength << " is not supported.");
    }
  }
  CUDA_CHECK_ERROR;

  // Cleanup
  for (unsigned int c = 0; c < vectorLength; c++)
  {
    hipFreeArray((hipArray *)volComponentArrays[c]);
    CUDA_CHECK_ERROR;
    hipDestroyTextureObject(tex_vol[c]);
    CUDA_CHECK_ERROR;
  }
  hipFree(dev_tex_vol);
  CUDA_CHECK_ERROR;
}
